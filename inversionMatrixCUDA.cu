#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#define MAX_DIM_ROW 1024
#define REPS 40

using namespace std;

int blocks;
int threads;
int n;

__global__ void inverseMatrixP1(double *M, double *X, double *R1, double *R2, int DIM)
{

    __shared__ double row[MAX_DIM_ROW];
    double sum;
    // Bucle que saltará numBlocks en cada iteración
    for (int d = blockIdx.x; d < DIM; d += gridDim.x)
    {

        // Bucle que con cada bloque usará hilos
        for (int i = threadIdx.x; i < DIM; i += blockDim.x)
        {

            // row será igual al valor de X en cierta posición.
            row[i] = *(X + i + DIM * d);
        }
        // Se sincronizan los hilos
        __syncthreads();

        // Se realizar el bucle para avanzar de acuero al id del hilo en el bloque por cada bloque
        for (int col = threadIdx.x; col < DIM; col += blockDim.x)
        {
            sum = 0;

            // Se recorre
            for (int pos = 0; pos < DIM; pos++)
            {

                // Se realiza la multiplicacion de las matrices teniendo en cuenta que la matriz X estará en row
                sum += row[pos] * *(M + col + DIM * pos);
            }

            // Se asigna a la matriz resultante el valor de la multiplicación
            *(R1 + col + DIM * d) = sum;
        }
        __syncthreads();
    }
}

__global__ void inverseMatrixP2(double *M, double *X, double *R1, double *R2, int DIM)
{

    __shared__ double row[MAX_DIM_ROW];
    double sum;
    for (int d = blockIdx.x; d < DIM; d += gridDim.x)
    {

        // Bucle que con cada bloque usará hilos
        for (int i = threadIdx.x; i < DIM; i += blockDim.x)
        {

            // row será igual al valor de A en cierta posición.
            row[i] = *(R1 + i + DIM * d);
        }
        // Se sincronizan los hilos
        __syncthreads();

        // Se realizar el bucle para avanzar de acuero al id del hilo en el bloque por cada bloque
        for (int col = threadIdx.x; col < DIM; col += blockDim.x)
        {
            sum = 0;

            // Se recorre
            for (int pos = 0; pos < DIM; pos++)
            {

                // Se realiza la multiplicacion de las matrices teniendo en cuenta que la matriz A estará en row
                sum += row[pos] * *(X + col + DIM * pos);
            }

            // Se asigna a la matriz resultante el valor de la multiplicación
            *(R2 + col + DIM * d) = 2 * *(X + col + DIM * d) - sum;
        }
        __syncthreads();
    }
}

__global__ void inverseMatrixP3(double *M, double *X, double *R1, double *R2, int DIM)
{
    for (int d = blockIdx.x; d < DIM; d += gridDim.x)
    {

        // Bucle que con cada bloque usará hilos
        for (int col = threadIdx.x; col < DIM; col += blockDim.x)
        {

            // row será igual al valor de A en cierta posición.
            *(X + col + DIM * d) = *(R2 + col + DIM * d);
        }
    }
    __syncthreads();
}

double *init_x(double *m)
{
    double max_column = -INT_MAX;
    double max_fila = -INT_MAX;
    for (int i = 0; i < n; i++)
    {
        double column = 0;
        double fila = 0;
        for (int j = 0; j < n; j++)
        {
            fila += abs(m[i * n + j]);
            column += abs(m[j * n + i]);
        }
        max_column = max(max_column, column);
        max_fila = max(max_fila, fila);
    }
    double *x = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            x[i * n + j] = m[j * n + i] / (max_column * max_fila);
        }
    }
    return x;
}
void print_matrix(double *a)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << a[i * n + j] << " ";
        }
        cout << endl;
    }
}

double matrix_diff(double *a, double *b)
{
    double ans = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            ans += abs(a[i * n + j] - b[i * n + j]);
        }
    }
    return ans;
}

void multiplication_matrix(double *a, double *b, double *resultado)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            resultado[i * n + j] = 0;
            for (int k = 0; k < n; k++)
            {
                resultado[i * n + j] += a[i * n + k] * b[k * n + j];
            }
        }
    }
}

/*****************************************************************************/
int calcInverse()
{

    hipError_t err = hipSuccess;
    cin >> n;
    double *matrix = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cin >> matrix[i * n + j];
        }
    }
    double *x = init_x(matrix);
    int matrix_size = (n * n) * sizeof(double);
    double *resultado = (double *)malloc(n * n * sizeof(double));

    double *cudaM;
    err = hipMalloc((void **)&cudaM, matrix_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *cudaR1;
    err = hipMalloc((void **)&cudaR1, matrix_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix R1(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *cudaR2;
    err = hipMalloc((void **)&cudaR2, matrix_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix R2(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *cudaX;
    err = hipMalloc((void **)&cudaX, matrix_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix X(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(cudaM, matrix, matrix_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to device matrix A(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(cudaX, x, matrix_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to device matrix A(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    const int cu_n = n;
    printf("Launching kernel %d, %d \n", blocks, threads);

    auto start = chrono::high_resolution_clock::now();

    for (int rep = 0; rep < REPS; rep++)
    {
        inverseMatrixP1<<<blocks, threads>>>(cudaM, cudaX, cudaR1, cudaR2, cu_n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipDeviceSynchronize();
        inverseMatrixP2<<<blocks, threads>>>(cudaM, cudaX, cudaR1, cudaR2, cu_n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipDeviceSynchronize();
        inverseMatrixP3<<<blocks, threads>>>(cudaM, cudaX, cudaR1, cudaR2, cu_n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipDeviceSynchronize();
    }

    inverseMatrixP1<<<blocks, threads>>>(cudaM, cudaX, cudaR1, cudaR2, cu_n);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    auto int_s = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "MatrixMult elapsed time is " << int_s.count() / (float)1000000 << " seconds " << endl;

    printf("Kernel finalizado\n");
    err = hipMemcpy(x, cudaX, matrix_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to device solution matrix(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(resultado, cudaR1, matrix_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy to device solution matrix(error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cudaM);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device original  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cudaX);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device solution (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cudaR1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device solution (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(cudaR2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device original  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    cout << endl;
    print_matrix(x);
    cout << endl;
    print_matrix(resultado);
    free(resultado);
    free(matrix);
    free(x);

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

int main(int argc, char *argv[])
{
    freopen("input 1000.txt", "r", stdin);
    freopen("output.txt", "w", stdout);
    if (argc != 3)
    {
        cout << "Usage: " << argv[0] << " <bloques> <hilos>" << endl;
        return 1;
    }
    blocks = atoi(argv[1]);
    threads = atoi(argv[2]);
    calcInverse();
}